#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int threadsPerBlock;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b, int threadsPerBlock)
{
    int i = threadIdx.x + blockIdx.x * threadsPerBlock;
    c[i] = a[i] + b[i];
}

__global__ void fillArrayKernel(int* a, int num, int threadsPerBlock) {
    int i = threadIdx.x + blockIdx.x * threadsPerBlock;
    int result = num + i;
    a[i] = result;
}

__global__ void printKernel(int* c, int threadsPerBlock) {
    int i = threadIdx.x + blockIdx.x * threadsPerBlock;
    printf("%d\n", i);
}

int main()
{
    const int arraySize = 5000;
    const int a[arraySize] = { 1 };
    const int b[arraySize] = { 10 };
    int c[arraySize] = { 0 };
    threadsPerBlock = 1000;
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_c, c, arraySize * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    fillArrayKernel << <5, threadsPerBlock >> > (dev_a, 10, threadsPerBlock);
    fillArrayKernel << <5, threadsPerBlock >> > (dev_b, 1, threadsPerBlock);

    hipDeviceSynchronize();

    hipEventRecord(start, 0);
    // <<< Number_of_blocks, Number_of_threads / block >>>
    addKernel <<<5, threadsPerBlock>>> (dev_c, dev_a, dev_b, threadsPerBlock);
    hipEventRecord(stop, 0);

    cudaStatus = hipDeviceSynchronize(); // Makes sure all threads finished.

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);


    printKernel<<<5, threadsPerBlock>>>(dev_c, threadsPerBlock);
    hipDeviceSynchronize();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost); // Copy result back from the GPUcudaMemcpy
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    
    
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    printf("Time elapsed the execution of Kernel: %f\n", elapsedTime);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   /* cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}
